#include "hip/hip_runtime.h"
//
// Created by zpx on 2023/06/03.
//

#include <glog/logging.h>
#include "ops/relu_op.h"
#include "layer/relu_layer.h"
#include "helper/cuda_helper.cu"

namespace kuiper_infer {
    __global__ void relu_kernel(float *input, float *output, int N) {
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx < N)output[idx] = MAX_CUDA(input[idx], 0.f);
    }

    void relu_gpu_kernel(const std::shared_ptr<Tensor<float>> &input, std::shared_ptr<Tensor<float>> &output) {
        float *dev_in, *dev_out;
        CUDA_CHECK(hipMalloc(&dev_in, sizeof(float) * input->size()));
        CUDA_CHECK(hipMalloc(&dev_out, sizeof(float) * input->size()));
        CUDA_CHECK(hipMemcpy(dev_in, input->data().memptr(), sizeof(float) * input->size(), hipMemcpyKind::hipMemcpyHostToDevice));
        uint32_t bs = 256 < input->size() ? 256 : input->size();
        uint32_t ceil = (input->size() + bs - 1) / bs;
        relu_kernel<<<ceil, bs>>>(dev_in, dev_out, input->size());
        CUDA_CHECK(hipMemcpy(output->data().memptr(), dev_out, sizeof(float) * output->size(), hipMemcpyKind::hipMemcpyDeviceToHost));
        CUDA_CHECK(hipFree(dev_in));
        CUDA_CHECK(hipFree(dev_out));
    }

    void ReluLayer::ForwardsCuda(const std::vector<std::shared_ptr<Tensor<float>>> &inputs,
                                 std::vector<std::shared_ptr<Tensor<float>>> &outputs) {
        CHECK(this->op_ != nullptr);
        CHECK(this->op_->op_type_ == OpType::kOperatorRelu);
        const uint32_t batch_size = inputs.size();
        CHECK(inputs.size() == outputs.size()) << "The input size not equal with output size";
        for (int i = 0; i < batch_size; i++) {
            CHECK(!inputs.at(i)->empty());
            outputs.at(i) = std::make_shared<ftensor>(inputs.at(i)->shapes());
            relu_gpu_kernel(inputs.at(i), outputs.at(i));
        }
    }
}