#include "hip/hip_runtime.h"
//
// Created by zpx on 2023/06/03.
//
#include <glog/logging.h>
#include "ops/hardSwish_op.h"
#include "layer/hardSwish_layer.h"
#include "helper/cuda_helper.cu"

namespace kuiper_infer {
    __global__ void hardSwishLayer_kernel(float *input, float *output, int N) {
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        if (index < N) {
            float tmp = input[index] + 3.0f;
            if (tmp < 0)tmp = 0.f;
            if (tmp > 6)tmp = 6.f;
            output[index] = input[index] * (tmp / 6.0f);
        }
    }

    void hardSwishLayer_gpu_kernel(const std::shared_ptr<Tensor<float>> &input, std::shared_ptr<Tensor<float>> &output) {
        float *dev_in, *dev_out;
        CUDA_CHECK(hipMalloc(&dev_in, sizeof(float) * input->size()));
        CUDA_CHECK(hipMalloc(&dev_out, sizeof(float) * input->size()));
        CUDA_CHECK(hipMemcpy(dev_in, input->data().memptr(), sizeof(float) * input->size(), hipMemcpyKind::hipMemcpyHostToDevice));
        uint32_t bs = 256 < input->size() ? 256 : input->size();
        uint32_t ceil = (input->size() + bs - 1) / bs;
        hardSwishLayer_kernel<<<ceil, bs>>>(dev_in, dev_out, input->size());
        CUDA_CHECK(hipMemcpy(output->data().memptr(), dev_out, sizeof(float) * output->size(), hipMemcpyKind::hipMemcpyDeviceToHost));
        CUDA_CHECK(hipFree(dev_in));
        CUDA_CHECK(hipFree(dev_out));
    }

    void HardSwishLayer::ForwardsCuda(const std::vector<std::shared_ptr<Tensor<float>>> &inputs,
                                      std::vector<std::shared_ptr<Tensor<float>>> &outputs) {
        CHECK(this->op_ != nullptr);
        CHECK(this->op_->op_type_ == OpType::kOperatorHardSwish);
        const uint32_t batch_size = inputs.size();
        CHECK(inputs.size() == outputs.size()) << "The input size not equal with output size";
        for (int i = 0; i < batch_size; i++) {
            CHECK(!inputs.at(i)->empty());
            outputs.at(i) = std::make_shared<ftensor>(inputs.at(i)->shapes());
            hardSwishLayer_gpu_kernel(inputs.at(i), outputs.at(i));
        }
    }
}