#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <armadillo>
#include "helper/cuda_helper.cu"

__global__ void fun(float *i, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N)printf("%f\n", i[index]);
}

int main() {
    arma::fmat A(5, 5, arma::fill::ones);
    float *a;
    hipMalloc(&a, sizeof(float) * A.size());
    hipMemcpy(a, A.memptr(), sizeof(float) * A.size(), hipMemcpyKind::hipMemcpyHostToDevice);
    fun<<<1, 25>>>(a, 25);
    hipDeviceSynchronize();
    return 0;
}
