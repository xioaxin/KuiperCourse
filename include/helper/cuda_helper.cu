#include <hip/hip_runtime.h>


#define CUDA_CHECK(call)                              \
do                                                    \
{                                                     \
    const hipError_t error_code = call;              \
    if (error_code != hipSuccess)                    \
    {                                                 \
        printf("CUDA Error:\n");                      \
        printf("    File:       %s\n", __FILE__);     \
        printf("    Line:       %d\n", __LINE__);     \
        printf("    Error code: %d\n", error_code);   \
        printf("    Error text: %s\n",                \
        hipGetErrorString(error_code));              \
        exit(1);                                      \
    }                                                 \
} while (0)

class TimeHelper {
public:
    static void init() {
        hipEventCreate(&start_);
        hipEventCreate(&end_);
    }

    static void TimeBegin() {
        hipEventRecord(start_);
    }

    static void TimeBegin(hipStream_t stream) {
        hipEventRecord(start_, stream);
    }

    static void TimeEnd(float *time) {
        hipEventRecord(end_);
        hipEventSynchronize(end_);
        hipEventElapsedTime(time, start_, end_);
    }

    static void TimeEnd(float *time, hipStream_t stream) {
        hipEventRecord(end_, stream);
        hipEventSynchronize(end_);
        hipEventElapsedTime(time, start_, end_);
    }

    static void destroy() {
        hipEventDestroy(start_);
        hipEventRecord(end_);
    }

private:
    static hipEvent_t start_, end_;
};

#define MAX_CUDA(a, b)(a)>=(b)?(a):(b)
#define MIN_CUDA(a, b)(a)<=(b)?(a):(b)
